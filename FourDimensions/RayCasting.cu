#include "hip/hip_runtime.h"
#include "RayCasting.cuh"
#include <execution>
#include <algorithm>
#include <thread>
#include <iostream>

#pragma region RayCaster
RayCaster::RayCaster(sf::RenderWindow& window, const Player& player, const std::vector<Visible*>& visibles, std::mutex& visiblesMutex, const unsigned height, const unsigned width, std::atomic<bool>& lockTest) :
	window(window), player(player), visibles(visibles), height(height), width(width), pixels(new sf::Uint8[width * height * 4]),
	distance(height * width), color(height * width), pVisiblesImage(nullptr)
{
	texture.create(width, height);
	sprite = sf::Sprite(texture);
}
RayCaster::~RayCaster()
{
	delete[] pixels;
}
void RayCaster::RayCastScreen()
{
	thrust::fill(distance.begin(), distance.end(), std::numeric_limits<FPN>::infinity());
	thrust::fill(color.begin(), color.end(), sf::Color::Blue.toInteger());
	for (Visible* visible : pVisiblesImage->visibles)
	{
		switch (visible->id)
		{
		case Space3D::id:
			thrust::for_each(
				thrust::make_zip_iterator(pixelIndex, distance.begin(), color.begin()),
				thrust::make_zip_iterator(pixelIndex + height * width, distance.end(), color.end()),
				thrust::make_zip_function(RayCasterSpace3DFunctor(pVisiblesImage->player, *static_cast<const Space3D*>(visible), height, width)));
			break;
		case Tetrahedron::id:
			thrust::for_each(
				thrust::make_zip_iterator(pixelIndex, distance.begin(), color.begin()),
				thrust::make_zip_iterator(pixelIndex + height * width, distance.end(), color.end()),
				thrust::make_zip_function(RayCasterTetrahedronFunctor(pVisiblesImage->player, *static_cast<const Tetrahedron*>(visible), height, width)));
			break;
		case Mesh::id:
			for (Tetrahedron tetrahedron : static_cast<const Mesh*>(visible)->absoluteTetrahedrons)
			{
				thrust::for_each(
					thrust::make_zip_iterator(pixelIndex, distance.begin(), color.begin()),
					thrust::make_zip_iterator(pixelIndex + height * width, distance.end(), color.end()),
					thrust::make_zip_function(RayCasterTetrahedronFunctor(pVisiblesImage->player, tetrahedron, height, width)));
			}
			break;
		}
	}
	thrust::host_vector<unsigned> hostColor = color;
	struct VertexArrayFiller
	{
		sf::Uint8* pixels;
		thrust::host_vector<unsigned>& colors;
		const unsigned shunkSize;
		VertexArrayFiller(sf::Uint8* const pixels, thrust::host_vector<unsigned>& colors, unsigned shunkSize) :
			pixels(pixels), colors(colors), shunkSize(shunkSize)
		{ }
		void operator()(unsigned i)
		{
			for (unsigned q = i * shunkSize; q < (i + 1) * shunkSize; q++)
			{
				sf::Color color(colors[q]);
				pixels[q * 4] = color.r;
				pixels[q * 4 + 1] = color.g;
				pixels[q * 4 + 2] = color.b;
				pixels[q * 4 + 3] = color.a;
			}
		}
	};
	std::vector<std::thread> threads;
	for (unsigned i = 0; i < 10; i++) {
		threads.push_back(std::thread(VertexArrayFiller{ pixels, hostColor, width * height / 10 }, i));
	}
	for (std::thread& thread : threads) {
		thread.join();
	}
	texture.update(pixels); //Very slow, but do not know how to make faster
	window.draw(sprite);
}
FPN RayCaster::RayCastCPU(Vector4 rayOrigin, Vector4 RayDirection)
{
	FPN closestDistance = std::numeric_limits<FPN>::infinity();
	for (Visible* visible : visibles)
	{
		FPN distance;
		switch (visible->id)
		{
		case Space3D::id:
			distance = static_cast<Space3D*>(visible)->RayCast(rayOrigin, RayDirection);
			break;
		case Tetrahedron::id:
			distance = static_cast<Tetrahedron*>(visible)->RayCast(rayOrigin, RayDirection);
			break;
		case Mesh::id:
			distance = static_cast<Mesh*>(visible)->RayCast(rayOrigin, RayDirection);
			break;
		}
		if (distance >= 0 && distance < closestDistance)
			closestDistance = distance;
	}
	return closestDistance;
}
#pragma endregion

#pragma region RayCasterFunctors
RayCasterFunctor::RayCasterFunctor(const Player player, const unsigned screenHeight, const unsigned screenWidth) :
	screenCenter(player.GetScreenCenter()), screenXVector(player.GetScreenXVector()), screenYVector(player.GetScreenYVector()),
	screenHeight(screenHeight), screenWidth(screenWidth)
{ }

#pragma region RayCasterSpace3DFunctor
RayCasterSpace3DFunctor::RayCasterSpace3DFunctor(
	const Player player, const Space3D space3D, const unsigned screenHeight, const unsigned screenWidth) :
	RayCasterFunctor(player, screenHeight, screenWidth), space3D(space3D), relativePositionTimesNorm((space3D.position - player.position) * space3D.norm)
{ }
void RayCasterSpace3DFunctor::operator()(const unsigned& i, FPN& distance, unsigned& color) const
{
	const unsigned x = i % screenWidth;
	const unsigned y = i / screenWidth;
	const FPN rayDirectionX = screenCenter.X +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.X +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.X;
	const FPN rayDirectionY = screenCenter.Y +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.Y +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.Y;
	const FPN rayDirectionZ = screenCenter.Z +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.Z +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.Z;
	const FPN rayDirectionW = screenCenter.W +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.W +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.W;

	const FPN currentDistance = relativePositionTimesNorm /
		(rayDirectionX * space3D.norm.X + rayDirectionY * space3D.norm.Y + rayDirectionZ * space3D.norm.Z + rayDirectionW * space3D.norm.W) *
		sqrt(rayDirectionX * rayDirectionX + rayDirectionY * rayDirectionY + rayDirectionZ * rayDirectionZ + rayDirectionW * rayDirectionW);
	if (currentDistance >= 0 && currentDistance < distance)
	{
		distance = currentDistance;
		color = space3D.color;
	}
}
#pragma endregion

#pragma region RayCasterTetrahedronFunctor
RayCasterTetrahedronFunctor::RayCasterTetrahedronFunctor(
	const Player player, const Tetrahedron tetrahedron, const unsigned screenHeight, const unsigned screenWidth) :
	RayCasterFunctor(player, screenHeight, screenWidth), tetrahedron(tetrahedron), p(player.position - tetrahedron.position)
{ }
//i is index converted to x and y. distance is shortet positive distance so far, and is replaced if better is found. color is
//best color so far, and is replaced by the color of this object if the distance to this is smaller. Strange things made to
//make it able to run on the GPU and fast.
void RayCasterTetrahedronFunctor::operator()(const unsigned& i, FPN& distance, unsigned& color) const
{
	const unsigned x = i % screenWidth;
	const unsigned y = i / screenWidth;
	const FPN rayDirectionX = screenCenter.X +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.X +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.X;
	const FPN rayDirectionY = screenCenter.Y +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.Y +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.Y;
	const FPN rayDirectionZ = screenCenter.Z +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.Z +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.Z;
	const FPN rayDirectionW = screenCenter.W +
		((FPN)x - (FPN)screenWidth / 2) / (FPN)screenHeight * screenXVector.W +
		((FPN)y - (FPN)screenHeight / 2) / (FPN)screenHeight * screenYVector.W;

	//const FPN a1 =
	//	p[0] * (c[][] * c[][] * v + c[][] * v * c[][] + v * c[][] * c[][]) +
	//	p[1] * (c[][] * c[][] * c[][] + c[][] * c[][] * c[][] + c[][] * c[][] * c[][]) +
	//	p[2] * (c[][] * c[][] * c[][] + c[][] * c[][] * c[][] + c[][] * c[][] * c[][]) +
	//	p[3] * (c[][] * c[][] * c[][] + c[][] * c[][] * c[][] + c[][] * c[][] * c[][]);
	//const FPN t = -(
	//	p[0] * (
	//		c[0][1] * c[1][2] * c[2][3] +
	//		c[][] * c[][] * c[][] +
	//		c[][] * c[][] * c[][]) +
	//	p[1] * (
	//		c[][] * c[][] * c[][] +
	//		c[][] * c[][] * c[][] +
	//		c[][] * c[][] * c[][]) +
	//	p[2] * (
	//		c[][] * c[][] * c[][] +
	//		c[][] * c[][] * c[][] +
	//		c[][] * c[][] * c[][]) +
	//	p[3] * (
	//		c[][] * c[][] * c[][] +
	//		c[][] * c[][] * c[][] +
	//		c[][] * c[][] * c[][]));

	FPN matrix[4][4] = {
		{tetrahedron.corners[0].X, tetrahedron.corners[1].X, tetrahedron.corners[2].X, rayDirectionX},
		{tetrahedron.corners[0].Y, tetrahedron.corners[1].Y, tetrahedron.corners[2].Y, rayDirectionY},
		{tetrahedron.corners[0].Z, tetrahedron.corners[1].Z, tetrahedron.corners[2].Z, rayDirectionZ},
		{tetrahedron.corners[0].W, tetrahedron.corners[1].W, tetrahedron.corners[2].W, rayDirectionW} };
	FPN result[4] = { p.X, p.Y, p.Z, p.W };

	//Make it lower triangular
	for (unsigned q = 3; q >= 1; q--)
	{
		if (matrix[q][q] < (FPN)1e-3 && matrix[q][q] > (FPN)-1e-3)
		{
			for (int i = q - 1; i >= 0; i--) //Should perhaps go throgh all potential rows to switch with and take best, instead of giving up if none good enough is found.
			{
				if (matrix[i][q] >= (FPN)1e-3 || matrix[i][q] <= (FPN)-1e-3)
				{
					//Switches rows i and q
					for (unsigned p = 0; p < 4; p++)
					{
						FPN temp1 = matrix[q][p];
						matrix[q][p] = matrix[i][p];
						matrix[i][p] = temp1;
					}
					FPN temp2 = result[q];
					result[q] = result[i];
					result[i] = temp2;
					break;
				}
			}
		}

		for (unsigned i = 0; i < q; i++)
		{
			FPN scalar = matrix[i][q] / matrix[q][q];
			//Subtract scalar * row q from row i
			for (unsigned p = 0; p < 4; p++)
			{
				matrix[i][p] -= scalar * matrix[q][p];
			}
			result[i] -= scalar * result[q];
		}
	}

	//Make all values in the diagonal 1
	for (unsigned i = 0; i < 4; i++)
	{
		FPN scalar = matrix[i][i];
		for (unsigned q = 0; q < 4; q++)
		{
			matrix[i][q] /= scalar;
		}
		result[i] /= scalar;
	}

	//Remove all entries below the diagonal
	for (unsigned i = 1; i < 4; i++)
	{
		for (unsigned q = 0; q < i; q++)
		{
			//Subtract clone.values[i][q] * row q from row i
			result[i] -= matrix[i][q] * result[q];
		}
	}

	FPN a1 = result[0];
	FPN a2 = result[1];
	FPN a3 = result[2];
	FPN currentDistance = -result[3] *
		sqrt(rayDirectionX * rayDirectionX + rayDirectionY * rayDirectionY + rayDirectionZ * rayDirectionZ + rayDirectionW * rayDirectionW);

	if (currentDistance >= 0 && currentDistance < distance &&
		a1 >= 0 && a2 >= 0 && a3 >= 0 && (
		!tetrahedron.actuallyParallelepiped && a1 + a2 + a3 <= 1 ||
		tetrahedron.actuallyParallelepiped && a1 <= 1 && a2 <= 1 && a3 <= 1))
	{
		distance = currentDistance;
		switch (tetrahedron.colorization.colorScheme)
		{
		case ColorSheme::simple:
			color = tetrahedron.colorization.color1;
			break;
		case ColorSheme::dragedCubes:
			if (a1 < (FPN)0.5 != a2 < (FPN)0.5 != a3 < (FPN)0.5)
				color = tetrahedron.colorization.color1;
			else
				color = tetrahedron.colorization.color2;
			break;
		case ColorSheme::gradual:
		{
			//color = (unsigned)(a1 * (FPN)tetrahedron.colorization.color1 + (1 - a1) * (FPN)tetrahedron.colorization.color2); //Essentially what we want but becomes strange with float and unsigned and different channels owerflowing
			unsigned color1Part = (unsigned)(a1 * 0x01000000u);
			unsigned color2Part = 0x01000000u - color1Part;
			color = 0x000000FFu |
				(color1Part * (tetrahedron.colorization.color1 >> 8 & 0x000000FFu) + color2Part * (tetrahedron.colorization.color2 >> 8 & 0x000000FFu) >> 16 & 0x0000FF00u) +
				(color1Part * (tetrahedron.colorization.color1 >> 16 & 0x000000FFu) + color2Part * (tetrahedron.colorization.color2 >> 16 & 0x000000FFu) >> 8 & 0x00FF0000u) +
				(color1Part * (tetrahedron.colorization.color1 >> 24 & 0x000000FFu) + color2Part * (tetrahedron.colorization.color2 >> 24 & 0x000000FFu) & 0xFF000000u);
		}
			break;
		default:
			color = 0xFFB6C1FF;
			break;
		}
	}
}
#pragma endregion
#pragma endregion
